#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/common_layers.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include <math.h> 

#include <iostream>
#include <fstream>
namespace caffe {



template <typename Dtype>
__global__ void multiplyAllChannelsByMaskGpu(const Dtype* blob, const Dtype*  mask_blob, int mask_num, Dtype* blob_result, int sz, const int blob_channels){
  int data_offset = 0;
  int mask_offset = mask_num * sz;

  CUDA_KERNEL_LOOP(index, blob_channels*sz){
    //for(int j = 0; j < blob_channels; j++){
      int j = index / sz;
      data_offset = j * sz;      
      
       blob_result[data_offset + index % sz] = mask_blob[mask_offset + index % sz] * blob[data_offset + index % sz];
     // caffe_gpu_mul(sz, blob + data_offset, mask_blob + mask_offset, blob_result + data_offset);
   // }
  } 
}


/*
template <typename Dtype>
void multiplyAllChannelsByMaskGpu(const Dtype* blob, const Dtype*  mask_blob, int mask_num, Dtype* blob_result, int sz, const int blob_channels){
  int data_offset = 0;
  int mask_offset = mask_num * sz;

  CUDA_KERNEL_LOOP(j, blob_channels){
    //for(int j = 0; j < blob_channels; j++){
      data_offset = j * sz;      
      caffe_gpu_mul(sz, blob + data_offset, mask_blob + mask_offset, blob_result + data_offset);
   // }
  } 
}
*/

template <typename Dtype>
void BilinearPatchLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {  

  for (int n = 0; n < bottom[0]->num(); n++){
    for (int i = 0; i < poolingFieldsNum; i++){
       multiplyAllChannelsByMaskGpu<<<CAFFE_GET_BLOCKS(bottom[0]->channels()*bottom[0]->height()*bottom[0]->width()), CAFFE_CUDA_NUM_THREADS>>>(bottom[0]->gpu_data() + bottom[0]->channels() * bottom[0]->height() * bottom[0]->width() * n, mask_buffer.gpu_data(), i, masked_buffer1.mutable_gpu_data(), bottom[0]->height()*bottom[0]->width(), bottom[0]->channels());

       multiplyAllChannelsByMaskGpu<<<CAFFE_GET_BLOCKS(bottom[1]->channels()*bottom[0]->height()*bottom[0]->width()), CAFFE_CUDA_NUM_THREADS>>>(bottom[1]->gpu_data() + bottom[1]->channels() * bottom[1]->height() * bottom[1]->width() * n, mask_buffer.gpu_data(), i, masked_buffer2.mutable_gpu_data(), bottom[1]->height()*bottom[1]->width(), bottom[1]->channels());

       caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, bottom[0]->channels(), bottom[1]->channels(), bottom[0]->height() * bottom[0]->width(),(Dtype)1., masked_buffer1.gpu_data(), masked_buffer2.gpu_data(), (Dtype)0., transpBuffer_top.mutable_gpu_data() + n * transpBuffer_top.channels()  + i * bottom[0]->channels() * bottom[1]->channels());

/*
caffe_gpu_geam_old<float>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N,
    const float alpha, const float* A, const float* B, const float beta,
    float* C) {
*/
       
    }

    caffe_gpu_geam_old(CblasNoTrans, CblasTrans, top[0]->channels(), top[0]->height() * top[0]->width(), (Dtype)0.0, top[0]->gpu_data() + n * top[0]->channels() * top[0]->width() * top[0]->height(), transpBuffer_top.gpu_data() + n * transpBuffer_top.channels(), (Dtype)1.0, top[0]->mutable_gpu_data() + n * top[0]->channels() * top[0]->width() * top[0]->height() );


  }
}

template <typename Dtype>
void BilinearPatchLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  caffe_gpu_set(bottom[0]->num()*bottom[0]->channels()*bottom[0]->height()*bottom[0]->width(), Dtype(0.0), bottom[0]->mutable_gpu_diff());
  caffe_gpu_set(bottom[1]->num()*bottom[1]->channels()*bottom[1]->height()*bottom[1]->width(), Dtype(0.0), bottom[1]->mutable_gpu_diff());


  for (int n = 0; n < bottom[0]->num(); n++){
  
    caffe_gpu_geam_old(CblasNoTrans, CblasTrans, top[0]->height() * top[0]->width(), top[0]->channels(), (Dtype)0.0, transpBuffer_top.gpu_diff() + n * transpBuffer_top.channels(),
top[0]->gpu_diff() + n * top[0]->channels() * top[0]->width() * top[0]->height(),  (Dtype)1.0, transpBuffer_top.mutable_gpu_diff() + n * transpBuffer_top.channels() );
    
    for(int i = 0; i < poolingFieldsNum; i++){
      if (propagate_down[0]) {
        
        multiplyAllChannelsByMaskGpu<<<CAFFE_GET_BLOCKS(bottom[1]->channels()*bottom[0]->height()*bottom[0]->width()), CAFFE_CUDA_NUM_THREADS>>>(bottom[1]->gpu_data() + bottom[1]->channels() * bottom[1]->height() * bottom[1]->width() * n, mask_buffer.gpu_data(), i, masked_buffer2.mutable_gpu_data(), bottom[1]->height()*bottom[1]->width(), bottom[1]->channels());

        caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, bottom[0]->channels(), bottom[0]->width()*bottom[0]->height(), bottom[1]->channels(),(Dtype)1., transpBuffer_top.gpu_diff() + n * transpBuffer_top.channels()  + i * bottom[0]->channels() * bottom[1]->channels(), masked_buffer2.gpu_data(), (Dtype)0., dlda_buffer.mutable_gpu_diff());
	
	
	multiplyAllChannelsByMaskGpu<<<CAFFE_GET_BLOCKS(bottom[0]->channels()*bottom[0]->height()*bottom[0]->width()), CAFFE_CUDA_NUM_THREADS>>>(dlda_buffer.gpu_diff(), mask_buffer.gpu_data(), i,dlda_buffer.mutable_gpu_diff(), bottom[0]->height()*bottom[0]->width(), bottom[0]->channels());

        caffe_gpu_add(bottom[0]->channels()*bottom[0]->height()*bottom[0]->width(), dlda_buffer.gpu_diff(), bottom[0]->gpu_diff() + bottom[0]->channels() * bottom[0]->height() * bottom[0]->width() * n, bottom[0]->mutable_gpu_diff() + bottom[0]->channels() * bottom[0]->height() * bottom[0]->width() * n);

      }
	
      if (propagate_down[1]) {

         multiplyAllChannelsByMaskGpu<<<CAFFE_GET_BLOCKS(bottom[0]->channels()*bottom[0]->height()*bottom[0]->width()), CAFFE_CUDA_NUM_THREADS>>>(bottom[0]->gpu_data() + bottom[0]->channels() * bottom[0]->height() * bottom[0]->width() * n, mask_buffer.gpu_data(), i, masked_buffer1.mutable_gpu_data(), bottom[0]->height()*bottom[0]->width(), bottom[0]->channels());
        
        caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans, bottom[1]->channels(), bottom[1]->width()*bottom[1]->height(), bottom[0]->channels(),(Dtype)1., transpBuffer_top.gpu_diff() + n * transpBuffer_top.channels()  + i * bottom[0]->channels() * bottom[1]->channels(), masked_buffer1.gpu_data(), (Dtype)0., dldb_buffer.mutable_gpu_diff());


	multiplyAllChannelsByMaskGpu<<<CAFFE_GET_BLOCKS(bottom[1]->channels()*bottom[0]->height()*bottom[0]->width()), CAFFE_CUDA_NUM_THREADS>>>(dldb_buffer.gpu_diff(),mask_buffer.gpu_data(), i,dldb_buffer.mutable_gpu_diff(), bottom[1]->height()*bottom[1]->width(), bottom[1]->channels());

        caffe_gpu_add(bottom[1]->channels()*bottom[1]->height()*bottom[1]->width(), dldb_buffer.gpu_diff(), bottom[1]->gpu_diff() + bottom[1]->channels() * bottom[1]->height() * bottom[1]->width() * n, bottom[1]->mutable_gpu_diff() + bottom[1]->channels() * bottom[1]->height() * bottom[1]->width() * n);

      }
    }
  }

}

INSTANTIATE_LAYER_GPU_FUNCS(BilinearPatchLayer);
}  // namespace caffe

