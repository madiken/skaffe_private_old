#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

#include "caffe/proto/caffe.pb.h"
#include "caffe/util/io.hpp"

namespace caffe {

template <typename Dtype>
__global__ void kernel_channel_sum(const int num, const int channels,
    const int spatial_dim, const Dtype* data, Dtype* channel_sum) {
  CUDA_KERNEL_LOOP(index, num * spatial_dim) {
    int n = index / spatial_dim;
    int s = index % spatial_dim;
    Dtype sum = 0;
    for (int c = 0; c < channels; ++c) {
      sum += data[(n * channels + c) * spatial_dim + s];
    }
    channel_sum[index] = sum;
  }
}

template <typename Dtype>
__global__ void EntropyLossForwardGPU(const int nthreads,
          const Dtype* prob_data, Dtype* loss) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    loss[index] = -prob_data[index] * log(max(prob_data[index], 
                                              Dtype(FLT_MIN)));
  }
}

template <typename Dtype>
void EntropyLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  softmax_layer_->Forward(softmax_bottom_vec_, softmax_top_vec_);
  const Dtype* prob_data = prob_.gpu_data();
  Dtype* entropy_data = entropy_.mutable_gpu_data();
  // Since this memory is not used for anything until it is overwritten
  // on the backward pass, we use it here to avoid having to allocate new GPU
  // memory to accumulate intermediate results in the kernel.
  Dtype* loss_data = bottom[0]->mutable_gpu_diff();
  const int nthreads = prob_.count();
  const int channels = bottom[0]->shape(softmax_axis_);

  // NOLINT_NEXT_LINE(whitespace/operators)
  EntropyLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, prob_data, loss_data);
  // Compute entropies for each pixel.
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_sum<Dtype><<<CAFFE_GET_BLOCKS(outer_num_ * inner_num_),
      CAFFE_CUDA_NUM_THREADS>>>(outer_num_, channels, inner_num_, loss_data,
      entropy_data);

  Dtype loss;
  caffe_gpu_asum(outer_num_ * inner_num_, entropy_data, &loss);
  loss /= outer_num_;
  top[0]->mutable_cpu_data()[0] = loss;

  // std::cout << "[DEBUG] " << loss << std::endl;

  if (top.size() == 2) {
    top[1]->ShareData(prob_);
  }
}

template <typename Dtype>
__global__ void EntropyLossBackwardGPU(const int nthreads, 
          const Dtype* prob_data, const Dtype* entropy_data, Dtype* bottom_diff, 
          const int spatial_dim) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
    const Dtype entropy_value = entropy_data[n];
    bottom_diff[index] = -prob_data[index] * 
                         (entropy_value + log(max(prob_data[index], 
                                                  Dtype(FLT_MIN))));
  }
}

template <typename Dtype>
void EntropyLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* prob_data = prob_.gpu_data();
    const Dtype* entropy_data = entropy_.gpu_data();
    const int nthreads = prob_.count();

    // NOLINT_NEXT_LINE(whitespace/operators)
    EntropyLossBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
        CAFFE_CUDA_NUM_THREADS>>>(nthreads, prob_data, entropy_data, 
        bottom_diff, inner_num_);

    const Dtype loss_weight = top[0]->cpu_diff()[0];
    caffe_gpu_scal(prob_.count(), loss_weight / outer_num_, bottom_diff);
  }

  // {
  //   shared_ptr<BlobProto> blobp(new BlobProto());
  //   bottom[0]->ToProto(blobp.get(), true);
  //   WriteProtoToBinaryFile(*blobp, "bottom.binaryproto");
  // }

  // exit(0);
}

INSTANTIATE_LAYER_GPU_FUNCS(EntropyLossLayer);

}  // namespace caffe
